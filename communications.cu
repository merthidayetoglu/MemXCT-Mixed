#include "vars.h"

extern int socketrayout;
extern int socketrayinc;
extern int *socketreduceout;
extern int *socketreduceinc;
extern int *socketreduceoutdispl;
extern int *socketreduceincdispl;
extern int *socketsendcomm;
extern int *socketrecvcomm;
extern int *socketsendcommdispl;
extern int *socketrecvcommdispl;
extern int *socketsendmap;
extern int *socketreducedispl;
extern int *socketreduceindex;
extern int *socketraydispl;
extern int *socketrayindex;
extern int *socketpackmap;
extern int *socketunpackmap;

extern int noderayout;
extern int noderayinc;
extern int *nodereduceout;
extern int *nodereduceinc;
extern int *nodereduceoutdispl;
extern int *nodereduceincdispl;
extern int *nodesendcomm;
extern int *noderecvcomm;
extern int *nodesendcommdispl;
extern int *noderecvcommdispl;
extern int *nodesendmap;
extern int *nodereducedispl;
extern int *nodereduceindex;
extern int *noderaydispl;
extern int *noderayindex;
extern int *nodepackmap;
extern int *nodeunpackmap;

extern int numthreads;
extern int numproc;
extern int myid;
extern MPI_Comm MPI_COMM_SOCKET;
extern int numproc_socket;
extern int myid_socket;
extern int numsocket;
extern MPI_Comm MPI_COMM_NODE;
extern int numproc_node;
extern int myid_node;
extern int numnode;

extern int *socketpackmap_d;
extern int *socketunpackmap_d;
extern int *socketreducedispl_d;
extern int *socketreduceindex_d;
extern int *nodepackmap_d;
extern int *nodeunpackmap_d;
extern int *nodereducedispl_d;
extern int *nodereduceindex_d;
extern int *raypackmap_d;
extern int *rayunpackmap_d;
extern int *noderaydispl_d;
extern int *noderayindex_d;

extern COMMPREC *socketreducesendbuff_d;
extern COMMPREC *socketreducerecvbuff_d;
extern COMMPREC *nodereducesendbuff_d;
extern COMMPREC *nodereducerecvbuff_d;
extern COMMPREC *nodesendbuff_d;
extern COMMPREC *noderecvbuff_d;
extern COMMPREC *nodesendbuff_h;
extern COMMPREC *noderecvbuff_h;

extern int numdevice;
extern int mydevice;

int *socketrecvbuffdispl_p;
COMMPREC **socketrecvbuff_p;
int *socketrecvdevice_p;
int *noderecvbuffdispl_p;
COMMPREC **noderecvbuff_p;
int *noderecvdevice_p;

void communications(){

  MPI_Request sendrequest[numproc];
  MPI_Request recvrequest[numproc];

  socketrecvbuff_p = new COMMPREC*[numproc_socket];
  socketrecvbuffdispl_p = new int[numproc_socket];
  hipIpcMemHandle_t sockethandle[numproc_socket];
  //RECEIVER SENDS MEMORY HANDLE
  for(int precv = 0; precv < numproc_socket; precv++)
    if(socketrecvcomm[precv]){
      MPI_Isend(&socketrecvcommdispl[precv],1,MPI_INT,precv,1,MPI_COMM_SOCKET,sendrequest+precv);
      if(myid_socket!=precv){
        hipIpcGetMemHandle(&sockethandle[precv],socketreducerecvbuff_d);
        MPI_Isend(&sockethandle[precv],sizeof(hipIpcMemHandle_t),MPI_BYTE,precv,0,MPI_COMM_SOCKET,recvrequest+precv);
      }
      else
        socketrecvbuff_p[precv] = socketreducerecvbuff_d;
    }
  //SENDER OPENS MEMORY HANDLE
  for(int psend = 0; psend < numproc_socket; psend++)
    if(socketsendcomm[psend]){
      MPI_Recv(&socketrecvbuffdispl_p[psend],1,MPI_INT,psend,1,MPI_COMM_SOCKET,MPI_STATUS_IGNORE);
      if(myid_socket!=psend){
        hipIpcMemHandle_t temphandle;
        MPI_Recv(&temphandle,sizeof(hipIpcMemHandle_t),MPI_BYTE,psend,0,MPI_COMM_SOCKET,MPI_STATUS_IGNORE);
        hipIpcOpenMemHandle((void**)&socketrecvbuff_p[psend],temphandle,hipIpcMemLazyEnablePeerAccess);
      }
    }
  //RECEIVER DEVICE ID
  socketrecvdevice_p = new int[numproc_socket];
  for(int p = 0; p < numproc_socket; p++)
    socketrecvdevice_p[p] = ((myid/numproc_socket)*numproc_socket+p)%numdevice;
  //SOCKET IPC WARM-UP
  {
    MPI_Barrier(MPI_COMM_SOCKET);
    double time = MPI_Wtime();
    for(int psend = 0; psend < numproc_socket; psend++){
      if(socketsendcomm[psend])
        hipMemcpyPeerAsync(socketrecvbuff_p[psend]+socketrecvbuffdispl_p[psend]*FFACTOR,socketrecvdevice_p[psend],socketreducesendbuff_d+socketsendcommdispl[psend]*FFACTOR,mydevice,sizeof(COMMPREC)*socketsendcomm[psend]*FFACTOR);
    }
    hipDeviceSynchronize();
    MPI_Barrier(MPI_COMM_SOCKET);
    if(myid==0)printf("socket warmup time %e\n",MPI_Wtime()-time);
  }
  noderecvbuff_p = new COMMPREC*[numproc_node];
  noderecvbuffdispl_p = new int[numproc_node];
  hipIpcMemHandle_t nodehandle[numproc_node];
  //RECEIVER SENDS MEMORY HANDLE
  for(int precv = 0; precv < numproc_node; precv++)
    if(noderecvcomm[precv]){
      MPI_Isend(&noderecvcommdispl[precv],1,MPI_INT,precv,1,MPI_COMM_NODE,sendrequest+precv);
      if(myid_node!=precv){
        hipIpcGetMemHandle(&nodehandle[precv],nodereducerecvbuff_d);
        MPI_Isend(&nodehandle[precv],sizeof(hipIpcMemHandle_t),MPI_BYTE,precv,0,MPI_COMM_NODE,recvrequest+precv);
      }
      else
        noderecvbuff_p[precv] = nodereducerecvbuff_d;
    }
  //RECEIVER SENDS MEMORY HANDLE
  for(int precv = 0; precv < numproc_node; precv++)
    if(noderecvcomm[precv]){
      MPI_Isend(&noderecvcommdispl[precv],1,MPI_INT,precv,1,MPI_COMM_NODE,sendrequest+precv);
      if(myid_node!=precv){
        hipIpcGetMemHandle(&nodehandle[precv],nodereducerecvbuff_d);
        MPI_Isend(&nodehandle[precv],sizeof(hipIpcMemHandle_t),MPI_BYTE,precv,0,MPI_COMM_NODE,recvrequest+precv);
      }
      else
        noderecvbuff_p[precv] = nodereducerecvbuff_d;
    }
  //SENDER OPENS MEMORY HANDLE
  for(int psend = 0; psend < numproc_node; psend++)
    if(nodesendcomm[psend]){
      MPI_Recv(&noderecvbuffdispl_p[psend],1,MPI_INT,psend,1,MPI_COMM_NODE,MPI_STATUS_IGNORE);
      if(myid_node!=psend){
        hipIpcMemHandle_t temphandle;
        MPI_Recv(&temphandle,sizeof(hipIpcMemHandle_t),MPI_BYTE,psend,0,MPI_COMM_NODE,MPI_STATUS_IGNORE);
        hipIpcOpenMemHandle((void**)&noderecvbuff_p[psend],temphandle,hipIpcMemLazyEnablePeerAccess);
      }
    }
  //RECEIVER DEVICE ID
  noderecvdevice_p = new int[numproc_node];
  for(int p = 0; p < numproc_node; p++)
    noderecvdevice_p[p] = ((myid/numproc_node)*numproc_node+p)%numdevice;
  //NODE IPC WARM-UP
  {
    MPI_Barrier(MPI_COMM_NODE);
    double time = MPI_Wtime();
    for(int psend = 0; psend < numproc_node; psend++){
      if(nodesendcomm[psend])
        hipMemcpyPeerAsync(noderecvbuff_p[psend]+noderecvbuffdispl_p[psend]*FFACTOR,noderecvdevice_p[psend],nodereducesendbuff_d+nodesendcommdispl[psend]*FFACTOR,mydevice,sizeof(COMMPREC)*nodesendcomm[psend]*FFACTOR);
    }
    hipDeviceSynchronize();
    MPI_Barrier(MPI_COMM_NODE);
    if(myid==0)printf("node warmup time %e\n",MPI_Wtime()-time);
  }
  //HOST IPC WARM-UP
  {
    MPI_Barrier(MPI_COMM_WORLD);
    double chtime = MPI_Wtime();
    {
      int sendcount = 0;
      int recvcount = 0;
      for(int p = 0; p < numproc; p++){
        if(nodereduceout[p]){
          MPI_Isend(nodesendbuff_h+nodereduceoutdispl[p]*FFACTOR,nodereduceout[p]*FFACTOR*sizeof(COMMPREC),MPI_BYTE,p,0,MPI_COMM_WORLD,sendrequest+sendcount);
          sendcount++;
        }
        if(nodereduceinc[p]){
          MPI_Irecv(noderecvbuff_h+nodereduceincdispl[p]*FFACTOR,nodereduceinc[p]*FFACTOR*sizeof(COMMPREC),MPI_BYTE,p,0,MPI_COMM_WORLD,recvrequest+recvcount);
          recvcount++;
        }
      }
      MPI_Waitall(recvcount,recvrequest,MPI_STATUSES_IGNORE);
    }
    MPI_Barrier(MPI_COMM_WORLD);
    if(myid==0)printf("rack time %e\n",MPI_Wtime()-chtime);
  }
}
